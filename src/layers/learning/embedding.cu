#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_EMBEDDING_LAYER_INSTANTIATE
#include "lbann/layers/learning/embedding.hpp"

namespace lbann {

namespace {

/** @brief Kernel for forward prop
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (embedding_dim / bsize) x mini_batch_size x 1
 */
template <typename TensorDataType>
__global__ void fp_kernel(El::Int num_embeddings,
                          El::Int embedding_dim,
                          El::Int mini_batch_size,
                          const TensorDataType* __restrict__ indices,
                          El::Int indices_stride,
                          const TensorDataType* __restrict__ embeddings,
                          El::Int embeddings_ldim,
                          TensorDataType* __restrict__ output,
                          El::Int output_ldim) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nthreadsy = blockDim.y * gridDim.y;
  for (El::Int j = gidy; j < mini_batch_size; j += nthreadsy) {
    const El::Int ind = static_cast<El::Int>(indices[j*indices_stride]);
    for (El::Int i = gidx; i < embedding_dim; i += nthreadsx) {
      auto& y = output[i+j*output_ldim];
      if (0 <= ind && ind < num_embeddings) {
        y = embeddings[i+ind*embeddings_ldim];
      }
      else {
        y = TensorDataType{0};
      }
    }
  }
}

/** @brief Kernel for backprop
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (embedding_dim / bsize) x mini_batch_size x 1
 */
template <typename TensorDataType>
__global__ void bp_kernel(El::Int num_embeddings,
                          El::Int embedding_dim,
                          El::Int mini_batch_size,
                          El::Int padding_idx,
                          const TensorDataType* __restrict__ indices,
                          El::Int indices_stride,
                          const TensorDataType* __restrict__ gradient_wrt_output,
                          El::Int gradient_wrt_output_ldim,
                          TensorDataType* __restrict__ gradient_wrt_embeddings,
                          El::Int gradient_wrt_embeddings_ldim) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nthreadsy = blockDim.y * gridDim.y;
  for (El::Int j = gidy; j < mini_batch_size; j += nthreadsy) {
    const El::Int ind = static_cast<El::Int>(indices[j*indices_stride]);
    if (0 <= ind && ind < num_embeddings && ind != padding_idx) {
      for (El::Int i = gidx; i < embedding_dim; i += nthreadsx) {
        const auto& dy = gradient_wrt_output[i+j*gradient_wrt_output_ldim];
        auto& dw = gradient_wrt_embeddings[i+ind*gradient_wrt_embeddings_ldim];
        cuda::atomic_add(&dw, dy);
      }
    }
  }
}

} // namespace

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void embedding_layer<TensorDataType, T_layout, Dev>::setup_matrices(const El::Grid& grid) {
  data_type_layer<TensorDataType>::setup_matrices(grid);
  this->m_gradient_wrt_embeddings.reset(new El::DistMatrix<TensorDataType, El::STAR, El::STAR, El::ELEMENT, El::Device::GPU>(grid));
}

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void embedding_layer<TensorDataType, T_layout, Dev>::fp_compute() {

  using GPUMatType = El::Matrix<TensorDataType, El::Device::GPU>;

  // Local data
  const auto& local_embeddings = dynamic_cast<const GPUMatType&>(this->get_data_type_weights(0).get_values().LockedMatrix());
  const auto& local_input = dynamic_cast<const GPUMatType&>(this->get_local_prev_activations());
  auto& local_output = dynamic_cast<GPUMatType&>(this->get_local_activations());

  // Launch CUDA kernel
  if (!local_input.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_output.Height() + block_size - 1) / block_size;
    grid_dims.y = local_output.Width();
    fp_kernel<<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
      this->m_num_embeddings,
      this->m_embedding_dim,
      local_input.Width(),
      local_input.LockedBuffer(),
      local_input.LDim(),
      local_embeddings.LockedBuffer(),
      local_embeddings.LDim(),
      local_output.Buffer(),
      local_output.LDim());
  }

}

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void embedding_layer<TensorDataType, T_layout, Dev>::bp_compute() {
  using GPUMatType = El::Matrix<TensorDataType, El::Device::GPU>;

  // Embedding layer is not differentiable w.r.t. inputs
  El::Zero(this->get_error_signals());

  // Nothing to be done if embeddings are not being optimized
  if (this->get_data_type_weights(0).get_optimizer() == nullptr) { return; }
  auto& opt = *this->get_data_type_weights(0).get_optimizer();

  // Local data
  const auto& local_input = dynamic_cast<const GPUMatType&>(this->get_local_prev_activations());
  auto& local_embedding_grad = dynamic_cast<GPUMatType&>(this->m_gradient_wrt_embeddings->Matrix());
  const auto& local_output_grad = dynamic_cast<const GPUMatType&>(this->get_local_prev_error_signals());

  // Launch CUDA kernel
  El::Zero(local_embedding_grad);
  if (!local_input.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_output_grad.Height() + block_size - 1) / block_size;
    grid_dims.y = local_output_grad.Width();
    bp_kernel<<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
      this->m_num_embeddings,
      this->m_embedding_dim,
      local_input.Width(),
      this->m_padding_idx,
      local_input.LockedBuffer(),
      local_input.LDim(),
      local_output_grad.LockedBuffer(),
      local_output_grad.LDim(),
      local_embedding_grad.Buffer(),
      local_embedding_grad.LDim());
  }
  opt.add_to_gradient(*this->m_gradient_wrt_embeddings, TensorDataType{1}, true);

}

// Explicit instantiation
template class embedding_layer<DataType, data_layout::DATA_PARALLEL, El::Device::GPU>;

} // namespace lbann
