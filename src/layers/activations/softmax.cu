#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_SOFTMAX_LAYER_INSTANTIATE
#include "lbann/layers/activations/softmax.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

<<<<<<< HEAD
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
/** Functor to ensure values are above threshold value */
struct threshold_op {
  __forceinline__ __device__ DataType operator()(const DataType& y) const {
    return cuda::max(y, cuda::sqrt(cuda::min<TensorDataType>()));
  }
};
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD

/** @brief Max functor */
template <class T>
struct max_op {
  __device__ __forceinline__
  DataType operator()(const T& x1, const T& x2) const {
    return cuda::max(x1, x2);
=======
/** Minimum output value to avoid denormalized floats. */
inline __device__ TensorDataType get_min_output() {
#ifdef LBANN_ENABLE_SOFTMAX_CUTOFF
  return cuda::sqrt(cuda::min<TensorDataType>());
#else
  return TensorDataType(0);
#endif // LBANN_ENABLE_SOFTMAX_CUTOFF
}

#ifdef LBANN_ENABLE_SOFTMAX_CUTOFF
/** Operator for thresholding output. */
struct fp_threshold_op {
  const TensorDataType min_output =this->get_min_output();
  inline __device__ TensorDataType operator()(const TensorDataType& y) const {
    return cuda::max(y, min_output);
  }
};
/** Operator for thresholding gradient w.r.t. input. */
struct bp_threshold_op {
  const TensorDataType min_output =this->get_min_output();
  inline __device__ TensorDataType operator()(const TensorDataType& y,
                                        const TensorDataType& dx) const {
    return (y > min_output) ? dx : TensorDataType(0);
>>>>>>> Working on the math and activations layers.
  }
};

/** @brief Kernel for max reduction on matrix columns
 *
 *  Each CUDA block computes the max over a subset of matrix entries
 *  and outputs the result. This is repeated multiple times for
 *  column-wise max reduction.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param values       (height x width) matrix
 *  @param max_values   (nblocksx x width) matrix
 */
<<<<<<< HEAD
template <size_t bsize>
__global__ void reduce_max_kernel(size_t height,
                                  size_t width,
                                  const DataType* __restrict__ values,
                                  size_t values_ldim,
                                  DataType* __restrict__ max_values) {
=======
template <El::Int block_size>
__global__ void reduce_max_kernel(El::Int height, El::Int width,
                                  const TensorDataType* __restrict__ values,
                                  El::Int values_ldim,
                                  TensorDataType* __restrict__ max_values) {
>>>>>>> Working on the math and activations layers.

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidx = blockIdx.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksx = gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Find largest value for each thread
<<<<<<< HEAD
    DataType thread_max_val{-cuda::infinity<DataType>()};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& val = values[row+col*values_ldim];
      thread_max_val = cuda::max(thread_max_val, val);
    }

    // Find largest value for each block
    const DataType block_max_val
      = cuda::block_reduce<bsize,1,1,DataType,max_op<DataType>>(thread_max_val);
=======
    TensorDataType private_max_val = -cuda::infinity<TensorDataType>();
    for (El::Int row = gidx; row < height; row += nthreadsx) {
      private_max_val = cuda::max(private_max_val,
                                  values[row + col * values_ldim]);
    }

    // Shared memory reduction to get largest value for each block
    __shared__ TensorDataType shared_max_vals[block_size];
    shared_max_vals[tid] = private_max_val;
    for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        shared_max_vals[tid] = cuda::max(shared_max_vals[tid],
                                         shared_max_vals[tid + stride]);
      }
    }
>>>>>>> Working on the math and activations layers.
    if (tid == 0) {
      max_values[bidx+col*nblocksx] = block_max_val;
    }

  }

}

/** @brief Compute exp(x-shift)
 *
 *  Also compute sum(exp(x-shift)) for each matrix column.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
<<<<<<< HEAD
template <size_t bsize>
__global__ void fp_exp_kernel(size_t height,
                              size_t width,
                              const DataType* __restrict__ input,
                              size_t input_ldim,
                              DataType* __restrict__ output,
                              size_t output_ldim,
                              const DataType* __restrict__ shifts,
                              DataType* __restrict__ sums) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {
    const auto& shift = shifts[col];

    // Exponentiate inputs and compute sum for each thread
    DataType thread_sum{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& x = input[row+col*input_ldim];
      auto& y = output[row+col*output_ldim];
      y = cuda::exp(x-shift);
      thread_sum += y;
=======
template <El::Int block_size>
__global__ void fp_exp_kernel(El::Int height, El::Int width,
                              const TensorDataType* __restrict__ input,
                              El::Int input_ldim,
                              TensorDataType* __restrict__ output,
                              El::Int output_ldim,
                              const TensorDataType* __restrict__ shifts,
                              El::Int shifts_stride,
                              TensorDataType* __restrict__ sums,
                              El::Int sums_stride) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nblocksy = gridDim.y;

  // Reduce each matrix column independently
  for (El::Int col = bidy; col < width; col += nblocksy) {
    const auto& shift = shifts[col * shifts_stride];

    // Exponentiate and compute sum for each thread
    TensorDataType private_sum = 0;
    for (El::Int row = gidx; row < height; row += nthreadsx) {
      const auto& x = input[row + col * input_ldim];
      auto& y = output[row + col * output_ldim];
      y = cuda::exp(x - shift);
      private_sum += y;
    }

    // Shared memory reduction to get sum for each block
    __shared__ TensorDataType shared_sums[block_size];
    shared_sums[tid] = private_sum;
    for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        shared_sums[tid] += shared_sums[tid + stride];
      }
>>>>>>> Working on the math and activations layers.
    }

    // Compute sum for each block
    const DataType block_sum = cuda::block_reduce<bsize,1,1>(thread_sum);
    if (tid == 0) {
      cuda::atomic_add(&sums[col], block_sum);
    }

  }

}

/** @brief Compute layer output
 *
 *  y = exp(x-shift) / sum(exp(x-shift))
 *
 *  If @c LBANN_ENABLE_SOFTMAX_THRESHOLD is set, small values are
 *  thresholded to a minimum value to avoid denormalized floats.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param output   On input, constains exp(x-shift). On output,
 *                  contains the layer output.
 *  @param sums     sum(exp(x-shift)) for each column
 */
<<<<<<< HEAD
__global__ void fp_output_kernel(size_t height,
                                 size_t width,
                                 DataType* __restrict__ output,
                                 size_t output_ldim,
                                 const DataType* __restrict__ sums) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& denom = sums[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      auto& y = output[row+col*output_ldim];
      y /= denom;
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
      y = cuda::max(y, cuda::sqrt(cuda::min<DataType>()));
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
=======
__global__ void fp_scale_kernel(El::Int height, El::Int width,
                                TensorDataType* __restrict__ output,
                                El::Int output_ldim,
                                const TensorDataType* __restrict__ sums,
                                El::Int sums_stride) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nblocksy = gridDim.y;
  const auto& min_output =this->get_min_output();
  for (El::Int col = bidy; col < width; col += nblocksy) {
    const auto& scale = 1 / sums[col * sums_stride];
    for (El::Int row = gidx; row < height; row += nthreadsx) {
      auto& y = output[row + col * output_ldim];
      y = cuda::max(scale * y, min_output);
>>>>>>> Working on the math and activations layers.
    }
  }
}

<<<<<<< HEAD
/** @brief Compute dot(y,dy) for each matrix column
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
template <size_t bsize>
__global__ void bp_dot_product_kernel(size_t height,
                                      size_t width,
                                      const DataType* __restrict__ output,
                                      size_t output_ldim,
                                      const DataType* __restrict__ gradient_wrt_output,
                                      size_t gradient_wrt_output_ldim,
                                      DataType* __restrict__ dot_products) {
=======
/** Compute dot products between output and gradient w.r.t. output. */
template <El::Int block_size>
__global__ void bp_dot_product_kernel(El::Int height, El::Int width,
                                      const TensorDataType* __restrict__ output,
                                      El::Int output_ldim,
                                      const TensorDataType* __restrict__ gradient_wrt_output,
                                      El::Int gradient_wrt_output_ldim,
                                      TensorDataType* __restrict__ dot_products,
                                      El::Int dot_products_stride) {
>>>>>>> Working on the math and activations layers.

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Compute dot product contribution for each thread
<<<<<<< HEAD
    DataType thread_dot_product{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      thread_dot_product += y * dy;
=======
    TensorDataType private_dot_product = 0;
    for (El::Int row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row + col * output_ldim];
      const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      private_dot_product += y * dy;
    }

    // Shared memory reduction to get contribution for each block
    __shared__ TensorDataType shared_dot_products[block_size];
    shared_dot_products[tid] = private_dot_product;
    for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        shared_dot_products[tid] += shared_dot_products[tid + stride];
      }
>>>>>>> Working on the math and activations layers.
    }

    // Compute dot product contribution for each block
    const DataType block_dot_product
      = cuda::block_reduce<bsize,1,1>(thread_dot_product);
    if (tid == 0) {
      cuda::atomic_add(&dot_products[col], block_dot_product);
    }

  }

}

<<<<<<< HEAD
/** @brief Compute gradient w.r.t. input
 *
 *  dx = y * (dy - dot(y,dy))
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param dot_products dot(y,dy) for each matrix column
 */
template <size_t bsize>
__global__ void bp_kernel(size_t height,
                          size_t width,
                          const DataType* __restrict__ output,
                          size_t output_ldim,
                          const DataType* __restrict__ gradient_wrt_output,
                          size_t gradient_wrt_output_ldim,
                          const DataType* __restrict__ dot_products,
                          DataType* __restrict__ gradient_wrt_input,
                          size_t gradient_wrt_input_ldim) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& y_dot_dy = dot_products[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row+col*gradient_wrt_input_ldim];
      dx = y * (dy - y_dot_dy);
=======
/** Compute gradient w.r.t. input. */
template <El::Int block_size>
__global__ void bp_kernel(El::Int height, El::Int width,
                          const TensorDataType* __restrict__ output,
                          El::Int output_ldim,
                          const TensorDataType* __restrict__ gradient_wrt_output,
                          El::Int gradient_wrt_output_ldim,
                          const TensorDataType* __restrict__ dot_products,
                          El::Int dot_products_stride,
                          TensorDataType* __restrict__ gradient_wrt_input,
                          El::Int gradient_wrt_input_ldim) {
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;
  const El::Int nthreadsx = blockDim.x * gridDim.x;
  const El::Int nblocksy = gridDim.y;
  const auto& min_output =this->get_min_output();
  for (El::Int col = bidy; col < width; col += nblocksy) {
    const auto& y_dot_dy = dot_products[col * dot_products_stride];
    for (El::Int row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row + col * output_ldim];
      const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = (y > min_output) ? y * (dy - y_dot_dy) : TensorDataType(0);
>>>>>>> Working on the math and activations layers.
    }
  }
}

} // namespace

template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::fp_compute() {
  constexpr TensorDataType zero = 0;
  constexpr TensorDataType one = 1;
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_prev_activations());
  auto& local_output = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_activations());
  if (!local_input.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxForward(cudnn::get_handle(),
                                    HIPDNN_SOFTMAX_ACCURATE,
                                    HIPDNN_SOFTMAX_MODE_INSTANCE,
                                    &one,
                                    m_tensors_cudnn_desc.get_prev_activations(),
                                    local_input.LockedBuffer(),
                                    &zero,
                                    m_tensors_cudnn_desc.get_activations(),
                                    local_output.Buffer()));
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
    cuda::apply_entrywise_unary_operator<threshold_op>(local_output,
                                                       local_output);
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
  }
}

template <>
void softmax_layer<data_layout::DATA_PARALLEL, El::Device::GPU>::bp_compute() {
  constexpr TensorDataType zero = 0;
  constexpr TensorDataType one = 1;
  const auto& local_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_error_signals());
  if (!local_output.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxBackward(cudnn::get_handle(),
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     HIPDNN_SOFTMAX_MODE_INSTANCE,
                                     &one,
                                     m_tensors_cudnn_desc.get_activations(),
                                     local_output.LockedBuffer(),
                                     m_tensors_cudnn_desc.get_prev_error_signals(),
                                     local_gradient_wrt_output.LockedBuffer(),
                                     &zero,
                                     m_tensors_cudnn_desc.get_error_signals(),
                                     local_gradient_wrt_input.Buffer()));
  }
}

template <>
void softmax_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>::fp_compute() {

  // Local matrices
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_prev_activations());
  auto& local_output = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_activations());
  auto& local_workspace = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(m_workspace->Matrix());
  const size_t local_height = local_input.Height();
  const size_t local_width = local_input.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Find max value in each column
  cuda::thrust::vector<TensorDataType> max_vals;
  if (local_output.IsEmpty()) {
    max_vals.resize(local_width,
                    -std::numeric_limits<TensorDataType>::infinity());
  }
  else {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    max_vals.resize(grid_dims.x * local_width);
    reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      max_vals.data().get());
    while (grid_dims.x > 1) {
      const size_t prev_height = grid_dims.x;
      grid_dims.x = (prev_height + block_size - 1) / block_size;
      cuda::thrust::vector<DataType> prev_vals(std::move(max_vals));
      max_vals.resize(grid_dims.x * local_width);
      reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
        prev_height, local_width,
        prev_vals.data().get(), prev_height,
        max_vals.data().get());
    }
  }
  El::mpi::AllReduce(max_vals.data().get(), max_vals.size(),
                     El::mpi::MAX, m_workspace->RedundantComm(),
                     sync_info);

  // Compute exp(x-max_val) and sum(exp(x-max_val))
  El::Zero(*m_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_exp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      local_output.Buffer(), local_output.LDim(),
      max_vals.data().get(),
      local_workspace.Buffer());
  }
  El::AllReduce(*m_workspace, m_workspace->RedundantComm());

  // Compute output
  // Note: y = exp(x-max_val) / sum(exp(x-max_val))
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_output_kernel<<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.Buffer(), local_output.LDim(),
      local_workspace.LockedBuffer());
  }

}

template <>
void softmax_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>::bp_compute() {

  // Local matrices
  const auto& local_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(this->get_local_error_signals());
  auto& local_workspace = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(m_workspace->Matrix());
  const auto& local_height = local_output.Height();
  const auto& local_width = local_output.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Compute dot(y,dy)
  El::Zero(local_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_dot_product_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        local_height, local_width,
        local_output.LockedBuffer(),
        local_output.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_workspace.Buffer());
  }
  El::AllReduce(*m_workspace, m_workspace->RedundantComm());

  // Compute gradient w.r.t. input
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.LockedBuffer(),
      local_output.LDim(),
      local_gradient_wrt_output.LockedBuffer(),
      local_gradient_wrt_output.LDim(),
      local_workspace.Buffer(),
      local_gradient_wrt_input.Buffer(),
      local_gradient_wrt_input.LDim());
  }

}

// Template instantiation
template class softmax_layer<
  float, data_layout::DATA_PARALLEL, El::Device::GPU>;
template class softmax_layer<
  float, data_layout::MODEL_PARALLEL, El::Device::GPU>;

} // namespace lbann
