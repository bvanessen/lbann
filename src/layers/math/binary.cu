#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_BINARY_LAYER_INSTANTIATE
#include "lbann/layers/math/binary.hpp"

namespace lbann {

namespace {

/** CUDA kernel to apply an binary backprop operator. */
template <typename TensorDataType, typename BinaryBackPropOperator>
__global__
void binary_backprop_operator_kernel(El::Int height, El::Int width,
                                     const TensorDataType* __restrict__ x1,
                                     El::Int x1_ldim,
                                     const TensorDataType* __restrict__ x2,
                                     El::Int x2_ldim,
                                     const TensorDataType* __restrict__ dy,
                                     El::Int dy_ldim,
                                     TensorDataType* __restrict__ dx1,
                                     El::Int dx1_ldim,
                                     TensorDataType* __restrict__ dx2,
                                     El::Int dx2_ldim) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  BinaryBackPropOperator op;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    op(x1[row + col * x1_ldim],
       x2[row + col * x2_ldim],
       dy[row + col * dy_ldim],
       dx1[row + col * dx1_ldim],
       dx2[row + col * dx2_ldim]);
  }
}


/** Apply a binary backprop operator to CPU data.
 *  The input and output data must be on CPU and must have the same
 *  dimensions. Given a binary function \f$ y = f(x_1,x_2) \f$, the
 *  corresponding BinaryBackPropOperator is a 5-ary function with the
 *  arguments \f$ x_1 \f$, \f$ x_2 \f$, \f$ dL/dy \f$, \f$ dL/dx_1\f$,
 *  \f$ dL/dx_2 \f$. The last two arguments should be overwritten when
 *  the BinaryBackPropOperator is called.
 */
template <typename TensorDataType, typename BinaryBackPropOperator>
void apply_binary_backprop_operator(const El::AbstractMatrix<TensorDataType>& x1,
                                    const El::AbstractMatrix<TensorDataType>& x2,
                                    const El::AbstractMatrix<TensorDataType>& dy,
                                    El::AbstractMatrix<TensorDataType>& dx1,
                                    El::AbstractMatrix<TensorDataType>& dx2) {

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  const El::Int height = x1.Height();
  const El::Int width = x1.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int)
      && grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch CUDA kernel
  if (grid_dim > 0) {
    CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
    binary_backprop_operator_kernel<BinaryBackPropOperator>
      <<<grid_dim, block_dim, 0, El::GPUManager::Stream()>>>(
        height, width,
        x1.LockedBuffer(), x1.LDim(),
        x2.LockedBuffer(), x2.LDim(),
        dy.LockedBuffer(), dy.LDim(),
        dx1.Buffer(), dx1.LDim(),
        dx2.Buffer(), dx2.LDim());
  }

}

// =========================================================
// Operator objects for entry-wise binary layers
// =========================================================
// Note: Binary operator corresponds to forward prop step
// (\f$ y = f(x_1,x_2) \f$) and 5-ary operator corresponds
// to back prop step
// (\f$ \frac{dL}{dx_i} = \frac{dL}{dy} \frac{df}{dx_i}(x_1,x_2) \f$).

/** Add operator. */
template <typename TensorDataType>
struct add_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 + x2;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy;
    dx2 = dy;
  }
};

/** Subtract operator. */
template <typename TensorDataType>
struct subtract_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 - x2;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy;
    dx2 = -dy;
  }
};

/** Multiply operator. */
template <typename TensorDataType>
struct multiply_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 * x2;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy * x2;
    dx2 = dy * x1;
  }
};

/** Divide operator. */
template <typename TensorDataType>
struct divide_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 / x2;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy / x2;
    dx2 = -dy * x1 / (x2*x2);
  }
};

/** Modulo operator. */
template <typename TensorDataType>
struct mod_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return cuda::mod(x1, x2);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy;
    dx2 = -dy * cuda::floor(x1 / x2);
  }
};

/** Power operator. */
template <typename TensorDataType>
struct pow_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return cuda::pow(x1, x2);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {

    dx1 = dy * x2 * cuda::pow(x1, x2 - TensorDataType(1));
    dx2 = dy * cuda::log(x1) * cuda::pow(x1, x2);
  }
};

/** Safe divide operator.
 *  If a standard division produces an infinity or NaN, zero is output
 *  instead.
 */
template <typename TensorDataType>
struct safe_divide_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& y = x1 / x2;
    if (isfinite(y)) { return y; }
    else             { return TensorDataType(0); }
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    const auto& y = x1 / x2;
    if (isfinite(y)) {
      dx1 = dy / x2;
      dx2 = -dy * x1 / (x2*x2);
    } else {
      dx1 = TensorDataType(0);
      dx2 = TensorDataType(0);
    }
  }
};

/** Squared difference operator. */
template <typename TensorDataType>
struct squared_difference_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& diff = x1 - x2;
    return diff * diff;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = dy * 2*(x1-x2);
    dx2 = dy * 2*(x2-x1);
  }
};

/** Maximum operator. */
template <typename TensorDataType>
struct max_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return cuda::max(x1, x2);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    if (x1 > x2) {
      dx1 = dy;
      dx2 = TensorDataType(0);
    } else if (x2 > x1) {
      dx1 = TensorDataType(0);
      dx2 = dy;
    } else {
      dx1 = dy / 2;
      dx2 = dy / 2;
    }
  }
};

/** Minimum operator. */
template <typename TensorDataType>
struct min_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return cuda::min(x1, x2);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    if (x1 < x2) {
      dx1 = dy;
      dx2 = TensorDataType(0);
    } else if (x2 < x1) {
      dx1 = TensorDataType(0);
      dx2 = dy;
    } else {
      dx1 = dy / 2;
      dx2 = dy / 2;
    }
  }
};

/** Equal operator. */
template <typename TensorDataType>
struct equal_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 == x2 ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Not equal operator. */
template <typename TensorDataType>
struct not_equal_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 == x2 ? TensorDataType(0) : TensorDataType(1);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Less than operator. */
template <typename TensorDataType>
struct less_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 < x2 ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Less than or equal operator. */
template <typename TensorDataType>
struct less_equal_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 <= x2 ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Greater than operator. */
template <typename TensorDataType>
struct greater_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 > x2 ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Greater than or equal operator. */
template <typename TensorDataType>
struct greater_equal_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    return x1 >= x2 ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Logical and operator. */
template <typename TensorDataType>
struct logical_and_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 != TensorDataType(0) && !isnan(x1);
    const auto& b2 = x2 != TensorDataType(0) && !isnan(x2);
    return (b1 && b2) ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Logical or operator. */
template <typename TensorDataType>
struct logical_or_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 != TensorDataType(0) && !isnan(x1);
    const auto& b2 = x2 != TensorDataType(0) && !isnan(x2);
    return (b1 || b2) ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

/** Logical xor operator. */
template <typename TensorDataType>
struct logical_xor_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 != TensorDataType(0) && !isnan(x1);
    const auto& b2 = x2 != TensorDataType(0) && !isnan(x2);
    return (b1 || b2) && !(b1 && b2) ? TensorDataType(1) : TensorDataType(0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0);
    dx2 = TensorDataType(0);
  }
};

} // namespace

// Template instantiation
#define INSTANTIATE(layer, op)                                                                   \
  template <typename TensorDataType>                                                             \
  void fp_compute_impl(layer<TensorDataType, data_layout::MODEL_PARALLEL, El::Device::GPU>& l) { \
    cuda::apply_entrywise_binary_operator<op<TensorDataType>>(l.get_prev_activations(0),         \
                                                              l.get_prev_activations(1),         \
                                                              l.get_activations());              \
  }                                                                                              \
  template <typename TensorDataType>                                                             \
  void bp_compute_impl(layer<TensorDataType, data_layout::MODEL_PARALLEL, El::Device::GPU>& l) { \
    apply_binary_backprop_operator<op<TensorDataType>>(l.get_local_prev_activations(0),          \
                                                       l.get_local_prev_activations(1),          \
                                                       l.get_local_prev_error_signals(),         \
                                                       l.get_local_error_signals(0),             \
                                                       l.get_local_error_signals(1));            \
  }                                                                                              \
  template <typename TensorDataType>                                                             \
  void fp_compute_impl(layer<TensorDataType, data_layout::DATA_PARALLEL, El::Device::GPU>& l) {  \
    cuda::apply_entrywise_binary_operator<op>(l.get_prev_activations(0),                         \
                                              l.get_prev_activations(1),                         \
                                              l.get_activations());                              \
  }                                                                                              \
  template <typename TensorDataType>                                                             \
  void bp_compute_impl(layer<TensorDataType, data_layout::DATA_PARALLEL, El::Device::GPU>& l) {  \
    apply_binary_backprop_operator<op>(l.get_local_prev_activations(0),                          \
                                       l.get_local_prev_activations(1),                          \
                                       l.get_local_prev_error_signals(),                         \
                                       l.get_local_error_signals(0),                             \
                                       l.get_local_error_signals(1));                            \
  }                                                                                              \
  BINARY_ETI_INST_MACRO_DEV(layer, El::Device::GPU)

INSTANTIATE(add_layer, add_op);
INSTANTIATE(subtract_layer, subtract_op);
INSTANTIATE(multiply_layer, multiply_op);
INSTANTIATE(divide_layer, divide_op);
INSTANTIATE(mod_layer, mod_op);
INSTANTIATE(pow_layer, pow_op);
INSTANTIATE(safe_divide_layer, safe_divide_op);
INSTANTIATE(squared_difference_layer, squared_difference_op);
INSTANTIATE(max_layer, max_op);
INSTANTIATE(min_layer, min_op);
INSTANTIATE(equal_layer, equal_op);
INSTANTIATE(not_equal_layer, not_equal_op);
INSTANTIATE(less_layer, less_op);
INSTANTIATE(less_equal_layer, less_equal_op);
INSTANTIATE(greater_layer, greater_op);
INSTANTIATE(greater_equal_layer, greater_equal_op);
INSTANTIATE(logical_and_layer, logical_and_op);
INSTANTIATE(logical_or_layer, logical_or_op);
INSTANTIATE(logical_xor_layer, logical_xor_op);

} // namespace lbann
